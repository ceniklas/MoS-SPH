#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <stdlib.h>
#include <crtdbg.h>
#include <GL/glew.h> // include GLEW and new version of GL on Windows
#include <glfw3.h> // GLFW helper library
#include <stdio.h>
#include <iostream>
#include <random>
#include <algorithm>

//#include "particles.h"
#include "shader.h"

#include <vector>
#include <math.h>
//#include <vld.h>

#define _CRTDBG_MAP_ALLOC

#define kParticlesCount 1024
#define kBorderSideParticlesCount 300
#define kBorderParticlesCount (4*kBorderSideParticlesCount)
#define kWindowWidth 640
#define kWindowHeight 480
#define kPi 3.14159265359
#define g -9.81
#define kFrameRate 60
#define kSubSteps 7

#define kOffset 0.90f

//#define kDt ((1.0f/kFrameRate) / kSubSteps)

#define averageParticles 20
//#define interactionRadius sqrt(averageParticles/(kParticlesCount*kPi))
#define interactionRadius 0.1f
#define IR2 interactionRadius*interactionRadius
#define cellSize (2.0f*interactionRadius)

void advance();
void render();
void glInit();
void particlesInit();
void borderParticlesInit();
void drawGrid();
void updateGrid();
void updatNeighbours();
void createDrawablePoints();
void calculatePressure();
void calulateForces();
float calculateMass();


#define kWdeafult (315/(64*kPi*pow(interactionRadius,9))); 
#define kWgradPressure -(45/(kPi*pow(interactionRadius,6)));
#define kWlaplacianViscosity (45/(kPi*pow(interactionRadius,6)));
#define kWgradDefult -(945/(32*kPi*pow(interactionRadius,9)));
#define kWlaplacianDefult -(945/(32*kPi*pow(interactionRadius,9)));

unsigned int vao;
unsigned int vbo;
unsigned int shader_programme;

const float kViewScale =  2.0f;
//const float interactionRadius =  0.05f;
//const float cellSize = 2*interactionRadius;



#define kDt  0.0005f;
const int kCellCount = 100;
const float restDensity = 988.0f;
const int kstiffnes = 20;
const float surfaceTension = 0.0728f;
const float viscosityConstant = 3.5f;

float particleMass;
float surfaceLimit = sqrt(restDensity/averageParticles);
float accelerationX;
float accelerationY;

//Every force
float   pressureForcex, pressureForcey, viscosityForcex,
        viscosityForcey,normalx, normaly, gradNormal,
        surfaceTensionForcex,surfaceTensionForcey, gravity;

float dx, dy, distance2;


struct particle
{
    float m_x;
    float m_y;
    float m_u; //x-velocity
    float m_v; //y-velocity

    float m_massDensity;
    float m_pressure;

    float m_mass;

    particle* next;
};

#define kMaxNeighbourCount 64
struct Neighbours
{
	//thrust::device_vector<particle*> particlesVec[kMaxNeighbourCount];
	const particle* particles[kMaxNeighbourCount];
    float r2[kMaxNeighbourCount];
    size_t count;
};

__shared__ Neighbours self_label;

struct point
{
    float x;
    float y;
};


particle particles[kParticlesCount];

particle borderParticles[kBorderParticlesCount];
const size_t kGridWidth = (size_t)(2.0 / cellSize);
const size_t kGridHeight = (size_t)(2.0 / cellSize);

const size_t kGridCellCount = kGridWidth * kGridHeight;
size_t gridCoords[kParticlesCount*2];
particle* gridArr[kGridCellCount];
std::vector<particle*> grid;

std::vector<point> drawablePoints;
std::vector<point> DEBUG_CORNER;

point acceleration[kParticlesCount];
point prevAcceleration[kParticlesCount];
float vhx[kParticlesCount];
float vhy[kParticlesCount];
bool firstIteration = true;

Neighbours neighbours[kParticlesCount];
GLuint programID = 0;

//Global CUDA arrays
__device__ particle  d_particles[kParticlesCount];
__device__ particle* d_grid[kGridCellCount];
thrust::device_vector<particle*> ddgrid;
 //thrust::device_vector<particle*> d_grid;
__device__ size_t d_gridCoords[2*kParticlesCount];
__device__ Neighbours d_neighbours[kParticlesCount];
__device__ particle d_borderParticles[kBorderParticlesCount];
__device__ float d_vhx[kParticlesCount];
__device__ float d_vhy[kParticlesCount]; 
__device__ bool d_firstIteration = true;

__global__ void updateGridDevice()
{
    const size_t d_kGridWidth = (size_t)(2.0 / cellSize);
    const size_t d_kGridHeight = (size_t)(2.0 / cellSize);
    const size_t d_kGridCellCount = d_kGridWidth * d_kGridHeight;
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    
        particle& pi = d_particles[i];

        int x = (1 + pi.m_x)/cellSize;
        int y = (1 + pi.m_y)/cellSize;


        if (x < 1)
            x = 1;
        else if (x > d_kGridWidth-2)
            x = d_kGridWidth-2;

        if (y < 1)
            y = 1;
        else if (y > d_kGridHeight-2)
            y = d_kGridHeight-2;

        pi.next = d_grid[x+y*d_kGridWidth];
        d_grid[x+y*d_kGridWidth] = &pi;

        d_gridCoords[i*2] = x;
        d_gridCoords[i*2+1] = y;
}

__global__ void updatNeighboursDevice()
{
	const size_t d_kGridWidth = (size_t)(2.0 / cellSize);
	int i = blockIdx.x * blockDim.x + threadIdx.x;

        particle& pi = d_particles[i];

        int x = (1 + pi.m_x)/cellSize;
        int y = (1 + pi.m_y)/cellSize;

        size_t gi = d_gridCoords[i*2];
        size_t gj = d_gridCoords[i*2+1]*d_kGridWidth;
        
        d_neighbours[i].count = 0;
        
        //Loop over border
        for(size_t j = 0; j < kBorderParticlesCount; j++)
        {
            particle bp = d_borderParticles[j];
            float pm = bp.m_mass;

            float dx = pi.m_x - bp.m_x;
            float dy = pi.m_y - bp.m_y;
            float distance2 = dx*dx + dy*dy;

            if(distance2 < IR2)
            {
                if(d_neighbours[i].count < kMaxNeighbourCount)
                {
                    d_neighbours[i].particles[d_neighbours[i].count] = &bp;
                    d_neighbours[i].r2[d_neighbours[i].count] = distance2;
                    ++d_neighbours[i].count;
                }
            }
        }
        //loop over cells
        for (int ni=gi-1; ni<=gi+1; ++ni)
        {
            for (int nj=gj-d_kGridWidth; nj<=gj+d_kGridWidth; nj+=d_kGridWidth)
            {
                //loop over neighbors
                for (particle* ppj=d_grid[ni+nj]; NULL!=ppj; ppj=ppj->next)
                {
                    //do fancy math
                    //std::cout << "ppj x: " << ppj->m_x << std::endl;
                    float dx = pi.m_x - ppj->m_x;
                    float dy = pi.m_y - ppj->m_y;
                    float distance2 = dx*dx + dy*dy;

                    if(distance2 < IR2)
                    {
                        //Density
                        //massDensity += particleMass*kWdeafult* (IR2 - distance2)*(IR2 - distance2)*(IR2 - distance2);

                        if(d_neighbours[i].count < kMaxNeighbourCount)
                        {
                            d_neighbours[i].particles[d_neighbours[i].count] = ppj;
                            d_neighbours[i].r2[d_neighbours[i].count] =distance2;
                            ++d_neighbours[i].count;
                        }
                    }
                }
            }
        }
}

__global__ void updateGridDevice2()
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	particle& pi = d_particles[i];

	const size_t d_kGridWidth = (size_t)(2.0 / cellSize);
    const size_t d_kGridHeight = (size_t)(2.0 / cellSize);
    const size_t d_kGridCellCount = d_kGridWidth * d_kGridHeight;

	int x = (1 + pi.m_x)/cellSize;
    int y = (1 + pi.m_y)/cellSize;

	if (x < 1)
            x = 1;
        else if (x > d_kGridWidth-2)
            x = d_kGridWidth-2;

	if (y < 1)
            y = 1;
        else if (y > d_kGridHeight-2)
            y = d_kGridHeight-2;

	pi.next = d_grid[x+y*d_kGridWidth];
    d_grid[x+y*d_kGridWidth] = &pi;

    d_gridCoords[i*2] = x;
    d_gridCoords[i*2+1] = y;

	//pi.m_u = d_gridCoords[i*2];
	//pi.m_v = d_gridCoords[i*2+1];

	pi.m_u = d_grid[x+y*d_kGridWidth]->m_mass;
	//pi.m_v = pi.next->m_mass;
}

__global__ void updateNeighboursDevice(){
	const size_t d_kGridWidth = (size_t)(2.0 / cellSize);
	int i = blockIdx.x * blockDim.x + threadIdx.x;

    particle& pi = d_particles[i];

	int x = (1 + pi.m_x)/cellSize;
	int y = (1 + pi.m_y)/cellSize;

	size_t gi = d_gridCoords[i*2];
	size_t gj = d_gridCoords[i*2]*d_kGridWidth;
	/*
	for(int ni = gi-1; ni<=gi+1; ++ni)
	{
		for(int nj = gj-d_kGridWidth; nj<=gj+d_kGridWidth; nj+=d_kGridWidth)
		{
			for(particle* ppj=d_grid[ni+nj]; NULL !=ppj; ppj=ppj->next)
			{

			}
		}
	}
	*/
	//particle* pj = d_grid[i];  
	pi.m_u = d_grid[i]->m_mass;
	//pi.m_v = 0.0f;
	//pi.m_v = gj;
}
__global__ void calculatePressureDevice(){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	particle& pi = d_particles[i];
	int count1 = 0;
	float massDensity = 0.0f;
	int count2 = 0;
	for(int j=0; j< kParticlesCount; j++){
		
		particle& pj = d_particles[j];
		float dx = pi.m_x - pj.m_x;
		float dy = pi.m_y - pj.m_y;
		float distance2 = dx*dx + dy*dy;
		float mass = pj.m_mass;
		
		if(distance2 < IR2){
			count1++;
			massDensity += mass*(IR2 - distance2)*(IR2 - distance2)*(IR2 - distance2)*kWdeafult;
			if(distance2 != 0){
				count2++;		
			}
		}
	}
	
	pi.m_massDensity = massDensity;
	pi.m_pressure = kstiffnes*(massDensity - 988);
	//pi.m_u = count1;
	//pi.m_v = count2;
}

__global__ void calclateForceDevice(){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	particle& pi = d_particles[i];
	
	float pressureForcex = 0.0f; 
    float pressureForcey = 0.0f;
    float viscosityForcex = 0.0f;
    float viscosityForcey = 0.0f;
    float normalx = 0.0f;
    float normaly = 0.0f;
    float gradNormal = 0.0f;
    float surfaceTensionForcex = 0.0f;
    float surfaceTensionForcey = 0.0f;

	float mdi = pi.m_massDensity;

	int count1 = 0;
	int count2 = 0;
	float k = 0.0f;
	for(int j=0; j< kParticlesCount; j++){
		
		particle& pj = d_particles[j];
		float dx = pi.m_x - pj.m_x;
		float dy = pi.m_y - pj.m_y;
		float distance2 = dx*dx + dy*dy;
		float mass = pj.m_mass;
		
		if(distance2 < IR2){
			float mdj = pj.m_massDensity;

			normalx += (mass/mdj)*(IR2-distance2)*(IR2-distance2)*dx*kWgradDefult;
            normaly += (mass/mdj)*(IR2-distance2)*(IR2-distance2)*dy*kWgradDefult;
			gradNormal += (mass/mdj)*(IR2-distance2)*(IR2-distance2)*(3*IR2-7*distance2)*kWlaplacianDefult;
			
			count1++;
			//massDensity += mass*(IR2 - distance2)*(IR2 - distance2)*(IR2 - distance2)*kWdeafult;
			if(distance2 != 0){
				float distance = sqrt(distance2);
				float velocityDiffu = pj.m_u - pi.m_u;
                float velocityDiffv = pj.m_v - pi.m_v;

				float pressi = pi.m_pressure;
				float pressj = pj.m_pressure;

				pressureForcex += ((pressi/(mdi*mdi))+(pressj/(mdj*mdj)))*mass*(interactionRadius-distance)*(interactionRadius-distance)*(dx/distance)*kWgradPressure;
                pressureForcey += ((pressi/(mdi*mdi))+(pressj/(mdj*mdj)))*mass*(interactionRadius-distance)*(interactionRadius-distance)*(dy/distance)*kWgradPressure;

				viscosityForcex += velocityDiffu * (mass/mdj) * (interactionRadius-distance)*kWlaplacianViscosity;
                viscosityForcey += velocityDiffv * (mass/mdj) * (interactionRadius-distance)*kWlaplacianViscosity;

			}
		}
	}

	float normalLenght = 1/sqrt(normalx*normalx + normaly*normaly);
    if(normalLenght > 7){
		surfaceTensionForcex = - 0.0728f  * gradNormal * normalx * normalLenght;
        surfaceTensionForcey = - 0.0728f  * gradNormal * normaly *normalLenght ;
    }

	pressureForcex = -mdi*pressureForcex;
	pressureForcey = -mdi*pressureForcey;

	viscosityForcex = 3.5f * viscosityForcex;
	viscosityForcey = 3.5 * viscosityForcey;

	float accX = (pressureForcex + viscosityForcex + surfaceTensionForcex)/mdi;
	float accY = ((pressureForcey + viscosityForcey + surfaceTensionForcey)/mdi);
	
	/*
	if(d_firstIteration){
		d_vhx[i] = pi.m_u + 0.5*accX*kDt;
        d_vhy[i] = pi.m_v + 0.5*accY*kDt;
                        
        pi.m_u += accX*kDt;
        pi.m_v += accY*kDt;
		
		pi.m_x += d_vhx[i]*kDt;
        pi.m_y += d_vhy[i]*kDt;

        d_firstIteration = false;
	}else{
		d_vhx[i] += accX*kDt;
        d_vhy[i] += accY*kDt;
		pi.m_u = d_vhx[i] + 0.5*accX*kDt;
		pi.m_v = d_vhy[i] + 0.5*accY*kDt;

		pi.m_x += d_vhx[i]*kDt;
		pi.m_y += d_vhy[i]*kDt;
	}
	*/
}


int main () {
  // start GL context and O/S window using the GLFW helper library
  if (!glfwInit ()) {
    fprintf (stderr, "ERROR: could not start GLFW3\n");
    return 1;
  } 


  GLFWwindow* window = glfwCreateWindow (kWindowWidth, kWindowHeight, "Here be fluids", NULL, NULL);
  if (!window) 
  {
    fprintf (stderr, "ERROR: could not open window with GLFW3\n");
    glfwTerminate();
    return 1;
  }
  glfwMakeContextCurrent (window);
                                  
  // start GLEW extension handler
  glewExperimental = GL_TRUE;
  glewInit ();

  // get version info
  const GLubyte* renderer = glGetString (GL_RENDERER); // get renderer string
  const GLubyte* version = glGetString (GL_VERSION); // version as a string
  printf ("Renderer: %s\n", renderer);
  printf ("OpenGL version supported %s\n", version);
  

  grid.resize(kGridCellCount);
  particlesInit();
  borderParticlesInit();
  updateGrid();
  drawablePoints.resize(kParticlesCount + kBorderParticlesCount);


  particleMass = calculateMass();
  std::cout << "Mass: "<< particleMass << std::endl;
  
  hipMemcpyToSymbol(HIP_SYMBOL(d_particles), particles, kParticlesCount*sizeof(particle));
  hipMemcpyToSymbol(HIP_SYMBOL(borderParticles), borderParticles, kBorderParticlesCount*sizeof(particle));
  //updateGridDevice<<< 1,kParticlesCount >>>();

  glInit();

  std::cout << "interaction radius: " << interactionRadius << std::endl;
  std::cout << "cellSize: " << cellSize << std::endl;
  std::cout << "grid cell count: " << kGridCellCount << std::endl;
  std::cout << "kgridwidth: " << kGridWidth << std::endl;
  //std::cout << "kDt: " << kDt << std::endl;


  double t = 0.0;
  double currentTime = glfwGetTime();
  double accumulator = 0.0;

  while (!glfwWindowShouldClose (window)) 
  {

	  /*for(int i = 0; i < kSubSteps; ++i)
      {
		updateGridDevice <<< 1, kParticlesCount >>>();
		updatNeighboursDevice <<< 1,kBorderParticlesCount >>> ();
		calculatePressureDevice<<< 16,64 >>>();
		calclateForceDevice<<<16, 64 >>>();
      }*/

	 updateGridDevice2<<< 1,kParticlesCount >>>();
	 //updateNeighboursDevice <<< 1,kParticlesCount>>>();

	  hipMemcpyFromSymbol(particles, HIP_SYMBOL(d_particles), kParticlesCount*sizeof(particle));

		for(int i=0; i<kParticlesCount; i++)
		{
			std::cout << "count1 = " << particles[i].m_u << std::endl;
			std::cout << "count2 = " << particles[i].m_v << std::endl;
		}
		
		
      createDrawablePoints();
      render();
	 

      glfwPollEvents ();
      // put the stuff we've been drawing onto the display
      glfwSwapBuffers (window);
  }


  // close GL context and any other GLFW resources
  glfwTerminate();
  return 0;
}

void glInit()
{
     programID = LoadShader( "default.vert", "flat.frag" );
    


    point topleft;
    point topright;
    point bottomleft;
    point bottomright;
    
    topleft.x = -1.0f;
    topleft.y = 1.0f;

    topright.x = 1.0f;
    topright.y = 1.0f;

    bottomleft.x = -1.0f;
    bottomleft.y = -1.0f;

    bottomright.x = 1.0f;
    bottomright.y = -1.0f;

    DEBUG_CORNER.push_back(topleft);
    DEBUG_CORNER.push_back(topright);
    DEBUG_CORNER.push_back(bottomleft);
    DEBUG_CORNER.push_back(bottomright);

    vbo = 0;
    glGenBuffers (1, &vbo);
    

    vao = 0;
    glGenVertexArrays (1, &vao);
    
    


}
void createDrawablePoints()
{
	
	//#pragma omp parallel for schedule(dynamic)
	for(int i = 0; i < kParticlesCount; ++i)
	{
		point p;
		p.x = particles[i].m_x;
		p.y = particles[i].m_y;
		drawablePoints[i] = p;
	}
	//#pragma omp parallel for schedule(dynamic)
	for(int i = 0; i < kBorderParticlesCount; ++i)
	{
		point p;
		p.x = borderParticles[i].m_x;
		p.y = borderParticles[i].m_y;
		drawablePoints[kParticlesCount + i] = p;
	}
}

void render()
{

    glClearColor(0.05f, 0.05f, 0.05f, 1);
	glClear (GL_COLOR_BUFFER_BIT);
	glUseProgram (programID);
    
	glBindBuffer (GL_ARRAY_BUFFER, vbo);
	glBufferData (GL_ARRAY_BUFFER, (kParticlesCount + kBorderParticlesCount) * sizeof(point), &drawablePoints[0], GL_STATIC_DRAW);
	glBindVertexArray (vao);
    //glBindVertexArray (vao);
	glEnableVertexAttribArray (0);
	glBindBuffer (GL_ARRAY_BUFFER, vbo);
	glVertexAttribPointer (0, 2, GL_FLOAT, GL_FALSE, 0, NULL);
    
 
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    glOrtho(0, 640, 0, 480, 0, 1);
 
	//Draw points as smooth balls (with AA)
    glEnable(GL_POINT_SMOOTH);
    glEnable(GL_BLEND);
    glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);

    glPointSize(5.0f);

    // draw points from the currently bound VAO with current in-use shader
    glDrawArrays (GL_POINTS, 0, kParticlesCount + kBorderParticlesCount);
	//glDisableVertexAttribArray(0);
}

void particlesInit()
{

	//std::mt19937 eng((std::random_device())());
	//std::uniform_real_distribution<> pos_dist(-0.001,0.001);

	int rowcolSize = sqrt(kParticlesCount);

  for(int particleIndexRow = 0; particleIndexRow < rowcolSize; ++particleIndexRow)
  {
	  float stepLength = 1.0f/rowcolSize;
	  for(int particleIndexCol = 0; particleIndexCol < rowcolSize; ++particleIndexCol)
	  {
		  particles[particleIndexCol + rowcolSize*particleIndexRow].m_x = -kOffset + particleIndexCol*stepLength; // + pos_dist(eng);
		  particles[particleIndexCol + rowcolSize*particleIndexRow].m_y = -kOffset + particleIndexRow*stepLength; // + pos_dist(eng);
	  }
  }
}

void borderParticlesInit()
{
	float stepLengthx = 2.0f/kBorderSideParticlesCount;
	float stepLengthy = 2.0f/kBorderSideParticlesCount;

	float mass = 0.1f;
	float md = 100.0f;
	float pressure = 0.01f;

	for(int i = 0; i < kBorderSideParticlesCount; i++)
	{
		borderParticles[i].m_x = -1.0f + stepLengthx*i;
		borderParticles[i].m_y = -0.98f;
		borderParticles[i].m_mass = mass;
		borderParticles[i].m_massDensity = md;
		borderParticles[i].m_pressure = pressure;
	}

	for(int i = 0; i < kBorderSideParticlesCount; i++)
	{
		borderParticles[kBorderSideParticlesCount + i].m_x = -0.98f;
		borderParticles[kBorderSideParticlesCount + i].m_y = (-1.0f) + stepLengthy*i;
		borderParticles[kBorderSideParticlesCount + i].m_mass = mass;
		borderParticles[kBorderSideParticlesCount + i].m_massDensity = md;
		borderParticles[kBorderSideParticlesCount + i].m_pressure = pressure;
	}

	for(int i = 0; i < kBorderSideParticlesCount; i++)
	{
		borderParticles[2*kBorderSideParticlesCount + i].m_x = 0.98f;
		borderParticles[2*kBorderSideParticlesCount + i].m_y = -1.0f + stepLengthy*i;
		borderParticles[2*kBorderSideParticlesCount + i].m_mass = mass;
		borderParticles[2*kBorderSideParticlesCount + i].m_massDensity = md;
		borderParticles[2*kBorderSideParticlesCount + i].m_pressure = pressure;
	}

	for(int i = 0; i < kBorderSideParticlesCount; i++)
	{
		borderParticles[3*kBorderSideParticlesCount + i].m_x = -1.0f + stepLengthx*i;
		borderParticles[3*kBorderSideParticlesCount + i].m_y = 0.98f;
		borderParticles[3*kBorderSideParticlesCount + i].m_mass = mass;
		borderParticles[3*kBorderSideParticlesCount + i].m_massDensity = md;
		borderParticles[3*kBorderSideParticlesCount + i].m_pressure = pressure;
	}

}

void updateGrid()
{
	memset(&grid[0], 0, kGridCellCount*sizeof(particle*));
	//grid.swap( std::vector<particle*>(grid.size(), 0) );
	for(size_t i = 0; i < kParticlesCount; i++)
	{
		particle& pi = particles[i];

		int x = (1 + pi.m_x)/cellSize;
		int y = (1 + pi.m_y)/cellSize;


		if (x < 1)
			x = 1;
		else if (x > kGridWidth-2)
			x = kGridWidth-2;

		if (y < 1)
			y = 1;
		else if (y > kGridHeight-2)
			y = kGridHeight-2;

		pi.next = grid[x+y*kGridWidth];
		grid[x+y*kGridWidth] = &pi;

		gridCoords[i*2] = x;
		gridCoords[i*2+1] = y;
	}
}

void updatNeighbours()
{
	for(int i = 0; i < kParticlesCount; ++i)
	{
		particle& pi = particles[i];

		int x = (1 + pi.m_x)/cellSize;
		int y = (1 + pi.m_y)/cellSize;

		size_t gi = gridCoords[i*2];
		size_t gj = gridCoords[i*2+1]*kGridWidth;
		
		neighbours[i].count = 0;
		
		//Loop over border
		for(size_t j = 0; j < kBorderParticlesCount; j++)
		{
			particle& bp = borderParticles[j];
			float pm = bp.m_mass;

			float dx = pi.m_x - bp.m_x;
			float dy = pi.m_y - bp.m_y;
			float distance2 = dx*dx + dy*dy;

			if(distance2 < IR2)
			{
				if(neighbours[i].count < kMaxNeighbourCount)
				{
					neighbours[i].particles[neighbours[i].count] = &bp;
					neighbours[i].r2[neighbours[i].count] = distance2;
					++neighbours[i].count;
					//std::cout << "I'm on the border" << std::endl;
				}
			}
		}
		//loop over cells
		for (int ni=gi-1; ni<=gi+1; ++ni)
		{
			for (int nj=gj-kGridWidth; nj<=gj+kGridWidth; nj+=kGridWidth)
			{
				//loop over neighbors
				for (particle* ppj=grid[ni+nj]; NULL!=ppj; ppj=ppj->next)
				{
					//do fancy math
					//std::cout << "ppj x: " << ppj->m_x << std::endl;
					dx = pi.m_x - ppj->m_x;
					dy = pi.m_y - ppj->m_y;
					distance2 = dx*dx + dy*dy;

					if(distance2 < IR2)
					{
						//Density
						//massDensity += particleMass*kWdeafult* (IR2 - distance2)*(IR2 - distance2)*(IR2 - distance2);

						if(neighbours[i].count < kMaxNeighbourCount)
						{
							neighbours[i].particles[neighbours[i].count] = ppj;
							neighbours[i].r2[neighbours[i].count] =distance2;
							++neighbours[i].count;
						}
					}
				}
			}
		}
	}
}


float calculateMass()
{
	float density = 0.0f; 
	

	for(int i = 0; i < kParticlesCount; ++i)
	{
		particle& pi = particles[i];

		int x = (1 + pi.m_x)/cellSize;
		int y = (1 + pi.m_y)/cellSize;

		size_t gi = gridCoords[i*2];
		size_t gj = gridCoords[i*2+1]*kGridWidth;

		//loop over cells 
		for (size_t ni=gi-1; ni<=gi+1; ++ni)
		{
			for (size_t nj=gj-kGridWidth; nj<=gj+kGridWidth; nj+=kGridWidth)
			{
				//loop over neighbors
				for (particle* ppj=grid[ni+nj]; NULL!=ppj; ppj=ppj->next)
				{
					const particle& pj = *ppj;
					dx = pi.m_x - pj.m_x;
					dy = pi.m_y - pj.m_y;
					distance2 = dx*dx + dy*dy;
					if(distance2 < IR2)
					{
						density += (IR2 - distance2)*(IR2 - distance2)*(IR2 - distance2)*kWdeafult;
					}
				}
			}
		}
	}
	float dA = density/kParticlesCount;
	float mass = (dA*restDensity)/(dA*dA);
	for(size_t i = 0; i < kParticlesCount; ++i)
	{
		particle& pi = particles[i];
		pi.m_mass = mass;
	}
	return mass;
}
